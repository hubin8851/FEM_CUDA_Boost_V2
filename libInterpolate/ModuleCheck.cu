#include <HbxDefMacro.h>
#include <libInterpolate\interval_lib.h>
//#include <libInterpolate\cuInterval_kernel.cuh>
#include <libInterpolate\spBaseLag.h>
#include <libInterpolate\interval_lib.h>

#include <hip/hip_cooperative_groups.h>

//CUDA��ֵ����У�����
template< unsigned int T>
bool check_interpolate( HBXDef::cuTable<T>* _table,
	HBXDef::cuLocation<T>* h_loc,
	HBXDef::UserCalPrec* h_result,
	unsigned int _num)
{
	//�ڴ˵���sp��CPU���ֵ��

	
	baselag<T>* thelag = new baselag<T>(_table, 0);

	for (unsigned int i = 0; i < _num; i++)
	{
		h_result[i] = thelag->get(h_loc[i].m_Loc);
	}



}