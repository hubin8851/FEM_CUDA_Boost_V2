
#include <hipblas.h>
#include <hipblas.h>
#include <hipblas.h>
#include "hip/hip_runtime_api.h"
#include <CuDefMacro.h>
#include <HbxCudaDef\AeroTable.h>
#include "cuCElastic.h"


//��ֵ����Xpos������
extern "C" 
void GetXposShape( HBXDef::UserCalPrec* _pSite, HBXDef::UserCalPrec* _pOutput )
{
	//���ÿ����Ĳ�ֵ���㲢���
}


//��ֵ����Xpos������б��
extern "C" 
void GetXposDShape( HBXDef::UserCalPrec* _pSite, HBXDef::UserCalPrec* _pOutput )
{
	//���ÿ����Ĳ�ֵ���㲢���
}

template<unsigned int _T, HBXDef::CudaMalloc_t _M>
cuCElastic<_T, _M>::cuCElastic()
{
	d_Cna = nullptr;		//����վ��������ϵ��
	d_Alpha = nullptr;		//���ع���
	d_Aero_Ni = nullptr;	//������ʱ��������
	isInit = false;

	m_cordlgth = 0;
	m_datalgth = 1;

	memset( m_lag_cordinate, 0, _T * sizeof(unsigned int) );
}

template<unsigned int _T, HBXDef::CudaMalloc_t _M>
cuCElastic<_T, _M>::~cuCElastic()
{
	delete[] d_Cna;
	delete[] d_Alpha;
	delete[] d_Aero_Ni;
}

//��ȡ��źͿ�ŵĹ��캯��
//@_IptTable:��ṹ��ָ��
//@_blkId:���
template<unsigned int _T, HBXDef::CudaMalloc_t _M>
cuCElastic<_T, _M>::cuCElastic(HBXDef::_AeroTable* _IptTable, size_t _blkId)
{
	Assert(_T>0)

	if (nullptr == _IptTable)
	{
		std::cerr << "�����������ݱ����..." << std::endl;
	}

	d_Cna = nullptr;		//����վ��������ϵ��
	d_Alpha = nullptr;		//���ع���
	d_Aero_Ni = nullptr;	//������ʱ��������

	m_cordlgth = 0;
	m_datalgth = 1;

	isInit = false;
	m_table_p = _IptTable;
	this->SetBlkId(_blkId);

	memset(m_lag_cordinate, 0, _T * sizeof(unsigned int));
}


//������캯��
template<unsigned int _T, HBXDef::CudaMalloc_t _M>
cuCElastic<_T, _M>::cuCElastic(const _sameCElastic& _rhs)
{
	Assert(_T>0)
	memcpy(m_numperdim, _rhs.m_numperdim, _T * sizeof(unsigned int));
	memset(m_lag_cordinate, 0, _T * sizeof(unsigned int));

	m_cordlgth = _rhs.m_cordlgth;
	m_datalgth = _rhs.m_datalgth;

	hipMemcpy( d_cordials, _rhs.d_cordials, m_cordlgth*sizeof(HBXDef::UserDefFloat), hipMemcpyDeviceToDevice );

	hipMemcpy(d_data, _rhs.d_data, m_datalgth*sizeof(HBXDef::UserDefFloat), hipMemcpyDeviceToDevice );


	isInit = true;
}


//@__blkId:��������block��ID,��0Ϊ��ʼ����
//@UserStatusError_t:����״̬������CheckUserDefErrors���ж�
template<unsigned int _T, HBXDef::CudaMalloc_t _M>
UserStatusError_t cuCElastic<_T, _M>::SetBlkId(size_t _blkId)
{
	if (_blkId >= m_table_p->_blocknum)
	{
		return UserStatusError_t::USER_STATUS_INVALID_VALUE;
	}
	m_slct_block_num = _blkId;
	m_dim = m_table_p->_blocks[_blkId]._dim;
	if (m_dim <= 0 || _T != m_dim)
	{
		return UserStatusError_t::USER_STATUS_INVALID_VALUE;
	}
	return UserStatusError_t::USER_STATUS_SUCCESS;
}

template<unsigned int _T, HBXDef::CudaMalloc_t _M>
void cuCElastic<_T, _M>::GenerateData()
{
	

#ifdef _CONST_MEM_	//ʹ�ó����ڴ�
	HBXDef::TrajInput	_tmpTraj;
	HBXDef::GenTraj(_tmpTraj);
	checkCudaErrors(hipMalloc((void**)&d_TrajInput_, sizeof(d_TrajInput_)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(&d_TrajInput_), &_tmpTraj, sizeof(d_TrajInput_), hipMemcpyHostToDevice));

	HBXDef::MonitorInput _tmpMonitor;
	HBXDef::GenMonitor(_tmpMonitor);
	checkCudaErrors(hipMalloc((void**)&d_Monitor_, sizeof(d_Monitor_)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(&d_Monitor_), &_tmpMonitor, sizeof(d_Monitor_), hipMemcpyHostToDevice));

	HBXDef::ShakeInput _tmpShake;
	HBXDef::GenShake(_tmpShake);
	checkCudaErrors(hipMalloc((void**)&d_ShakeInput_, sizeof(d_ShakeInput_)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(&d_ShakeInput_), &_tmpShake, sizeof(d_ShakeInput_), hipMemcpyHostToDevice));

	HBXDef::MassInput _tmpMass;
	HBXDef::GenMassInput(_tmpMass, nsMax);
	checkCudaErrors(hipMalloc((void**)&d_MassInput_, sizeof(d_MassInput_)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(&d_MassInput_), &_tmpMass, sizeof(d_MassInput_), hipMemcpyHostToDevice));
#else

	HBXDef::TrajInput	_tmpTraj;
	HBXDef::GenTraj(_tmpTraj);
	checkCudaErrors( hipMalloc( (void**)&d_TrajInput_, sizeof(d_TrajInput_)) );
	checkCudaErrors( hipMemcpy( &d_TrajInput_, &_tmpTraj, sizeof(d_TrajInput_), hipMemcpyHostToDevice ) );

	HBXDef::MonitorInput _tmpMonitor;
	HBXDef::GenMonitor(_tmpMonitor);
	checkCudaErrors(hipMalloc((void**)&d_Monitor_, sizeof(d_Monitor_)));
	checkCudaErrors(hipMemcpy(&d_Monitor_, &_tmpMonitor, sizeof(d_Monitor_), hipMemcpyHostToDevice));

	HBXDef::ShakeInput _tmpShake;
	HBXDef::GenShake(_tmpShake);
	checkCudaErrors(hipMalloc((void**)&d_ShakeInput_, sizeof(d_ShakeInput_)));
	checkCudaErrors(hipMemcpy(&d_ShakeInput_, &_tmpShake, sizeof(d_ShakeInput_), hipMemcpyHostToDevice));

	HBXDef::MassInput _tmpMass;
	HBXDef::GenMassInput(_tmpMass, nsMax);
	checkCudaErrors(hipMalloc((void**)&d_MassInput_, sizeof(d_MassInput_)));
	checkCudaErrors(hipMemcpy(&d_MassInput_, &_tmpMass, sizeof(d_MassInput_), hipMemcpyHostToDevice));

#endif // _CONST_MEM_
}


//��ʼ��������ɲ�ֵ�����ݵĴ��룬�Դ�ķ����
//@_DataIn:��ǰ���´����ֵ�����ݣ�����ģ�������GetParamģ���д���
template<unsigned int _T, HBXDef::CudaMalloc_t _M>
void cuCElastic<_T, _M>::Initial( void* _DataIn, hipStream_t* _stream )
{
	using namespace HBXDef;
	if (true == isInit)
	{
		std::cout << "�ڴ�֮ǰ�Ѿ�����˳�ʼ��" << std::endl;
		return;
	}

	IFNULL(_DataIn, "����ģ���ʼ��ʱ�����ָ��...");

	_plan = (HBXDef::OptionPlan*)_DataIn;
//	m_table_p = (HBXDef::OptionPlan*)_DataIn;//��ȡ�������ݱ�ָ��

	this->SetBlkId(0);	//Ŀǰ��ֵΪ0����Ϊÿ��table��ֻ��һ��block

//	memcpy( m_numperdim, m_table_p->_blocks[m_slct_block_num]._numperdim, sizeof(unsigned int) * _T );
	memcpy(m_numperdim, _plan->_pAeroBlock->_numperdim, sizeof(unsigned int) * _T);

	for (size_t i = 0; i < m_dim; i++)
	{
		m_cordlgth += m_numperdim[i];
		m_datalgth *= m_numperdim[i];
	}
	//�ڴ��ж����ݵĴ�С�Ƿ���Ϲ����ڴ������
	if (49152 < 4*m_datalgth)
	{
		printf("�޷�ʹ�ù����ڴ�...����ֵ��Ĵ�С�����˹����ڴ��48K����...");
 	}

	size_t d_pitchBytes;	//���Ե�ַ����
	size_t h_pitchBytes = m_numperdim[0] * sizeof(UserDefFloat);	//case Ϊ pitch ʱʹ�� Error���ڴ�����δ�е�ַ����
	hipChannelFormatDesc _channelDesc;	//case Ϊ array ʱʹ��
	switch (_M)
	{
	case HBXDef::CudaMalloc_t::PAGELOCK:	//ҳ�����ڴ�������첽����
		checkCudaErrors( hipMalloc((void**)&d_cordials, m_cordlgth * sizeof(UserDefFloat)) );	//Ϊ��������ڴ�
		checkCudaErrors( hipMalloc((void**)&d_data, m_datalgth * sizeof(UserDefFloat)) );	//Ϊ���ݷ����ڴ�
//		checkCudaErrors(hipMemcpyAsync(d_cordials, _plan->_pAeroBlock._corddata, sizeof(UserDefFloat)*m_cordlgth, hipMemcpyHostToDevice, *_stream));
//		checkCudaErrors(hipMemcpyAsync(d_data, _plan->_pAeroBlock._data, sizeof(UserDefFloat)*m_datalgth, hipMemcpyHostToDevice, *_stream));
		checkCudaErrors( hipMemcpyAsync(d_cordials, m_table_p->_blocks[m_slct_block_num]._corddata, sizeof(UserDefFloat)*m_cordlgth, hipMemcpyHostToDevice, *_stream) );
		checkCudaErrors( hipMemcpyAsync(d_data, m_table_p->_blocks[m_slct_block_num]._data, sizeof(UserDefFloat)*m_datalgth, hipMemcpyHostToDevice, *_stream));
		break;
	case HBXDef::CudaMalloc_t::NORMAL: default:	//������ʽ���ڴ����
		checkCudaErrors(hipMalloc((void**)&d_cordials, m_cordlgth * sizeof(UserDefFloat)));	//Ϊ��������ڴ�
		checkCudaErrors(hipMalloc((void**)&d_data, m_datalgth * sizeof(UserDefFloat)));	//Ϊ���ݷ����ڴ�
		checkCudaErrors(hipMemcpy(d_cordials, m_table_p->_blocks[m_slct_block_num]._corddata, sizeof(UserDefFloat)*m_cordlgth, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_data, m_table_p->_blocks[m_slct_block_num]._data, sizeof(UserDefFloat)*m_datalgth, hipMemcpyHostToDevice));
		break;
	case HBXDef::CudaMalloc_t::ZEROCPY:		//�㿽��ģʽ,����GPU�˷����Դ�
#if CUDART_VERSION >= 2020	//ֻ�д��ڵ���2.2���豸����ʵ��...
		checkCudaErrors( hipHostGetDevicePointer((void**)&d_cordials, (void*)m_table_p->_blocks[m_slct_block_num]._corddata, 0) );
		checkCudaErrors( hipHostGetDevicePointer((void**)&d_data, (void*)m_table_p->_blocks[m_slct_block_num]._data, 0) );
#endif
		break;
	case HBXDef::CudaMalloc_t::PITCH:	//��ַ������ʽ�������ڿ��ٲ���
		if (2 == m_dim)
		{
			checkCudaErrors( hipMallocPitch((void**)&d_cordials, &d_pitchBytes, m_numperdim[0] * sizeof(UserDefFloat), m_numperdim[1]) );
			checkCudaErrors( hipMemcpy2D( d_cordials, d_pitchBytes, m_table_p->_blocks[m_slct_block_num]._corddata, h_pitchBytes, m_numperdim[0] * sizeof(UserDefFloat), m_numperdim[1], hipMemcpyHostToDevice) );
			checkCudaErrors( hipMallocPitch((void**)&d_data, &d_pitchBytes, m_numperdim[0] * sizeof(UserDefFloat), m_numperdim[1]));
		}
		else if (3 == m_dim)
		{
			checkCudaErrors(hipMallocPitch((void**)&d_cordials, &d_pitchBytes, m_numperdim[0] * m_numperdim[1] * sizeof(UserDefFloat), m_numperdim[2]));
			printf("ֻ��ά��Ϊ2�ĸ�����õ�ַ���뷽ʽ�洢����ǰ��ά��Ϊ3...���ֽ�1-2ά�ϲ�Ϊ1ά");
		}
		else std::cerr << "����ά��Ŀǰ�в�����..." << std::endl;
		break;
	case HBXDef::CudaMalloc_t::ARRAY2D:
		if (2 == m_dim)
		{
			_channelDesc = hipCreateChannelDesc<UserDefFloat>();
			checkCudaErrors(hipMallocArray(&d_cordialsArray, &_channelDesc, m_numperdim[0], m_numperdim[1]));
			checkCudaErrors(hipMemcpy2DToArray(d_cordialsArray, 0, 0,	//dst array��offset
							m_table_p->_blocks[m_slct_block_num]._corddata, m_numperdim[0] * sizeof(UserDefFloat),	//src,pitch 
							m_numperdim[0] * sizeof(UserDefFloat), m_numperdim[1], hipMemcpyHostToDevice));	//��͸�
		}
		else if (3 == m_dim)
		{
			_channelDesc = hipCreateChannelDesc<UserDefFloat>();
			checkCudaErrors(hipMallocArray(&d_cordialsArray, &_channelDesc, m_numperdim[0]* m_numperdim[1], m_numperdim[2]));
			checkCudaErrors(hipMemcpy2DToArray(d_cordialsArray, 0, 0,	//dst array��offset
				m_table_p->_blocks[m_slct_block_num]._corddata, m_numperdim[0] * m_numperdim[1] * sizeof(UserDefFloat),	//src,pitch 
				m_numperdim[0] * m_numperdim[1] * sizeof(UserDefFloat), m_numperdim[2], hipMemcpyHostToDevice));	//��͸�
			printf("ֻ��ά��Ϊ2�ĸ�����õ�ַ���뷽ʽ�洢����ǰ��ά��Ϊ3...���ֽ�1-2ά�ϲ�Ϊ1ά");
		}
		else std::cerr << "����ά��Ŀǰ�в�����..." << std::endl;
		break;
	}

//	_cordlgth = 0;	//�ڴ�����άתһά�������ʱ����
//	for (int i = 0; i < m_dim; i++)
//	{
//		for (int j = 0; j < m_numperdim[i]; j++)
//		{
//			m_cordials[_cordlgth] = m_table_p->_blocks[m_slct_block_num]._corddata[i][j];
//			_cordlgth++;
//		}
//	}

	//���ͺ�����б�ʷ����ڴ档
	checkCudaErrors(hipMalloc((void**)&d_ModeData_.ShapeStationElastic, g_nE * g_nStation*sizeof(hbxFloat)) );
	checkCudaErrors(hipMalloc((void**)&d_ModeData_.DShapeStationElastic, g_nE * g_nStation*sizeof(hbxFloat)) );

	checkCudaErrors( hipMalloc((void**)&m_Interoplate, sizeof(hbxFloat)*(g_nStation+8)) );	//

//	cuElastic_Aero_Force << <16, 16 >> >(1, this->d_TrajInput_, this->d_ModeData_, this->d_MassInput_, 100);
	isInit = true;
}


//��ȡÿһ���㲽��������
template<unsigned int _T, HBXDef::CudaMalloc_t _M>
void cuCElastic<_T, _M>::getdata( void* _dataStep, hipStream_t* _stream )
{
	if (nullptr == _dataStep || false == isInit)
	{
		this->GenerateData();
	}
	

}

//�ú�����ȡ���нڵ㣨������վ�㣩�ڸ���ģ̬��ֵ���ò����������ڲ�����������ֱ���㣬����ʹ������20170626
template<unsigned int _T, HBXDef::CudaMalloc_t _M>
void cuCElastic<_T, _M>::GetInterpolatePointParam()
{
//	cubaselag<_T><<<>>>()
}

//���ǵ��Ա���֮�������������,20170626,�˺��Զ��������ʽ�ĳ��첽����
//template<unsigned int _T>
//void CElastic<_T>::Elastic_Aero_Force( hipblasHandle_t handle )
//{
//	hipblasStatus_t	_errorcode;
//	HBXDef::UserDefFloat _Alpha[nsMax];	//	����վ��ֲ�����
////	HBXDef::UserDefFloat* d_Qs;
//	HBXDef::UserDefFloat* d_Qsl;
//	HBXDef::UserDefFloat* d_Df_q;
//	HBXDef::UserDefFloat* d_f_Dq;
//
//	hipMalloc((void**)&d_Df_q, sizeof(HBXDef::UserDefFloat)* nsMax);
//	hipMalloc((void**)&d_f_Dq, sizeof(HBXDef::UserDefFloat)* nsMax);
//
//	float const_one = 1.0;
//	float const_DEG = 57.4;
//	float const_nDEG = -57.4;
//
//#ifdef _DOUBLEPRECISION_
//	//˫���ȵ�CUBLAS�汾��Ŀǰ��δ��д��20170626
//#endif
//	_errorcode = hipblasSscal( handle, nsMax, &d_TrajInput_.Q, d_ModeData_.S, 1);	//��ʱS��������Q���Qs����
//
//	/* ����ֲ����ǺͲ໬�� */
//	GetXposShape( d_MassInput_.xStation, d_ModeData_.qi );	//��������վ�������
////	_errorcode = hipblasSasum( handle, );
//	GetXposDShape( d_MassInput_.xStation, d_ModeData_.Dqi );	//��������վ�������б��
////	_errorcode = hipblasSasum(handle, );
//	_errorcode = hipblasSaxpy( handle, nsMax, &const_one, d_Df_q, 1, d_f_Dq, 1 );
//	_errorcode = hipblasSscal( handle, nsMax, &const_DEG, d_f_Dq, 1 );
////����һ���Ӹ�������to be continue
//
//	// ����ֲ��������ͷֲ��������� 
//	_errorcode = hipblasSdot( handle, nsMax, d_Cna, 1, d_Alpha, 1, d_Aero_Ni);	// �ֲ������� !!!
//	_errorcode = hipblasSdot( handle, nsMax, d_ModeData_.S, 1, d_Aero_Ni, 1, d_Aero_Ni);
////	_errorcode = hipblasSscal( handle, nsMax, &Qs, d_Aero_Ni, 1 );	//���QsΪ���������øú���
//}


//����������
template<unsigned int _T, HBXDef::CudaMalloc_t _M>
void cuCElastic<_T, _M>::Gen_Force_Cal()
{

}

//GPU����ӿ�,��ȱ
template<unsigned int _T, HBXDef::CudaMalloc_t _M>
void cuCElastic<_T, _M>::CalculateGPU(hipStream_t _stream)
{

}