#include "hip/hip_runtime.h"
#include <libInterpolate\cuInterpolate.h>
#include <libInterpolate\interval_lib.h>
#include <libInterpolate\cuInterval_kernel.cuh>
#include <libInterpolate\spBaseLag.h>
#include <hip/hip_runtime_api.h>
#include <time.h>

//#include <thrust\host_vector.h>//���ܵ���NVCC����������
//#include <thrust\transform.h>

//�ú�������ɺ˺����ⲿ�ĳ����Ż������漰�ڴ���������
//T����ʾά��
//M����ʾmalloc�ķ�ʽ
//@_table:��ֵ��
//@_posIn������ֵ������
//@_DataOut:��ֵ������ֵ����
template< unsigned int T>
float cuInterpolate(HBXDef::cuTable<T>* _table,
					HBXDef::cuLocation<T>* _posIn,
					HBXDef::UserCalPrec* _DataOut,
					unsigned int N)
{
	using namespace  HBXDef;
	using namespace  HBXDef::HbxCuDef;

	checkCudaErrors(hipSetDevice(0));

	float elapsedTime;	//����ʱ��

	//�鿴�ñ�Ĵ�С�Ա����shared memory�Ĵ�С
	if ( _table->size() < 1024*16 )
	{
		//ƫ����L1 32k���档��ΪL1�����ڿ��Է�����ص���ʱ����
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuInter_kernel<T>), hipFuncCachePreferL1);
	}
	else if( _table->size()< 1024* 32 && _table->size() > 1024 * 16)
	{
		//32-32���档��ΪL1�����ڿ��Է�����ص���ʱ����
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuInter_kernel<T>), hipFuncCachePreferEqual);
	}
	else if (_table->size()< 1024 * 48 && _table->size() > 1024 * 32)
	{
		//16-48���档��ΪL1�����ڿ��Է�����ص���ʱ����
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuInter_kernel<T>), hipFuncCachePreferShared);
	}

	//��֤ջ���㹻�Ŀռ��ŷ���������
//	hipDeviceSetLimit(hipLimitStackSize, 8192);

	hipEvent_t start, stop;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	checkCudaErrors(hipEventRecord(start, 0));//��ʼ��ʱ


	//�˺�����important
	cuInter_kernel<T> << < GRIDSIZE, BLOCKSIZE, MAX_SHARED, 0 >> > (_table, _posIn, _DataOut, N);
//	cuInter_TEST<T> << < GRIDSIZE, BLOCKSIZE, MAX_SHARED, 0 >> > (_table, _posIn, _DataOut);

	hipEventRecord(stop, 0);	//��ʱ����
	hipDeviceSynchronize();	//�߳�ͬ��
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);//��ȡ��ʱ

//	std::cout << "�˺���ִ��ʱ��:" << elapsedTime << std::endl;

	return elapsedTime;
};

//CUDA��ֵ����У�����
template< unsigned int T>
float Interpolate_cpu(HBXDef::cuTable<T>* _CuTable,
	HBXDef::cuLocation<T>* h_loc,
	HBXDef::UserCalPrec* h_result,
	unsigned int _num)
{
	//�ڴ˵���sp��CPU���ֵ��


	baselag<T>* thelag = new baselag<T>(_CuTable, 0);

	clock_t start, end;
	double duration;
	start = clock();

	float rsum = 0.0;
	float err = 0.0;
	for (unsigned int i = 0; i < _num; i++)
	{
		float diff = 0;
		h_result[i] = thelag->get(h_loc[i].m_Loc);
	}

	end = clock();

	duration = (double)(end - start) / CLOCKS_PER_SEC;

	std::cout << "���а�ִ��ʱ��:" << duration << std::endl;

	return duration;
}


#ifdef __cplusplus
	extern "C" {
#endif

#ifdef __cplusplus
}
#endif