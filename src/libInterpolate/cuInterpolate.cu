#include <libInterpolate\cuInterpolate.h>
#include <libInterpolate\interval_lib.h>
#include <libInterpolate\cuInterval_kernel.cuh>


//@T����ʾά��
//@M����ʾmalloc�ķ�ʽ
template< unsigned int T>
float cuInterpolate(HBXDef::cuTable<T>* _table,
					HBXDef::cuLocation<T>* _posIn,
					HBXDef::UserCalPrec* _DataOut)
{
	using namespace  HBXDef;
	using namespace  HBXDef::HbxCuDef;

	float elapsedTime;//����ʱ��

	//�鿴�ñ�Ĵ�С�Ա����shared memory�Ĵ�С
	if ( _table->size() < 1024*16 )
	{
		//ƫ����L1���档��ΪL1�����ڿ��Է�����ص���ʱ����
//		hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuInter_kernel<T>), hipFuncCachePreferL1);
	}
	else if( _table->size()< 1024* 32 && _table->size() > 1024 * 16)
	{
		//ƫ����L1���档��ΪL1�����ڿ��Է�����ص���ʱ����
//		hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuInter_kernel<T>), hipFuncCachePreferEqual);
	}
	else if (_table->size()< 1024 * 48 && _table->size() > 1024 * 32)
	{
		//ƫ����L1���档��ΪL1�����ڿ��Է�����ص���ʱ����
//		hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuInter_kernel<T>), cudaFuncCachePrefershared);
	}

	//��֤ջ���㹻�Ŀռ��ŷ���������
	hipDeviceSetLimit(hipLimitStackSize, 8192);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);//��ʼ��ʱ

	//�˺�����important
	cuInter_kernel<T> << < GRIDSIZE, BLOCKSIZE, MAX_SHARED, 0 >> > (_table, _posIn, _DataOut);

	hipEventRecord(stop, 0);	//��ʱ����
	hipDeviceSynchronize();	//�߳�ͬ��
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);//��ȡ��ʱ

//	std::cout << "time eclaps:" << elapsedTime << std::endl;

	return elapsedTime;
};


#ifdef __cplusplus
	extern "C" {
#endif

#ifdef __cplusplus
}
#endif