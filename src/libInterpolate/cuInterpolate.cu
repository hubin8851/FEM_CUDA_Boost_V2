#include <libInterpolate\cuInterpolate.h>
#include <libInterpolate\interval_lib.h>
#include <libInterpolate\cuInterval_kernel.cuh>
#include <hip/hip_runtime_api.h>

//T����ʾά��
//M����ʾmalloc�ķ�ʽ
//@_table:��ֵ��
//@_posIn������ֵ������
//@_DataOut:��ֵ������ֵ����
template< unsigned int T>
float cuInterpolate(HBXDef::cuTable<T>* _table,
					HBXDef::cuLocation<T>* _posIn,
					HBXDef::UserCalPrec* _DataOut)
{
	using namespace  HBXDef;
	using namespace  HBXDef::HbxCuDef;

	float elapsedTime;//����ʱ��


	//�鿴�ñ�Ĵ�С�Ա����shared memory�Ĵ�С
	if ( _table->size() < 1024*16 )
	{
		//ƫ����L1 32k���档��ΪL1�����ڿ��Է�����ص���ʱ����
//		hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuInter_kernel<T>), hipFuncCachePreferL1);
	}
	else if( _table->size()< 1024* 32 && _table->size() > 1024 * 16)
	{
		//32-32���档��ΪL1�����ڿ��Է�����ص���ʱ����
//		hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuInter_kernel<T>), hipFuncCachePreferEqual);
	}
	else if (_table->size()< 1024 * 48 && _table->size() > 1024 * 32)
	{
		//16-48���档��ΪL1�����ڿ��Է�����ص���ʱ����
//		hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuInter_kernel<T>), hipFuncCachePreferShared);
	}

	//��֤ջ���㹻�Ŀռ��ŷ���������
//	hipDeviceSetLimit(hipLimitStackSize, 8192);

	checkCudaErrors(hipSetDevice(0));

	hipEvent_t start, stop;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	checkCudaErrors(hipEventRecord(start, 0));//��ʼ��ʱ

	//�˺�����important
	cuInter_kernel<T> << < GRIDSIZE, BLOCKSIZE, MAX_SHARED, 0 >> > (_table, _posIn, _DataOut);
//	cuInter_TEST<T> << < GRIDSIZE, BLOCKSIZE, MAX_SHARED, 0 >> > (_table, _posIn, _DataOut);

	hipEventRecord(stop, 0);	//��ʱ����
	hipDeviceSynchronize();	//�߳�ͬ��
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);//��ȡ��ʱ

	std::cout << "time eclaps:" << elapsedTime << std::endl;

	return elapsedTime;
};


#ifdef __cplusplus
	extern "C" {
#endif

#ifdef __cplusplus
}
#endif