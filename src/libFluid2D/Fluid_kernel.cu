#include "hip/hip_runtime.h"
#include <CudaPreDef.h>
#include <hipfft/hipfft.h>
#include <helper_gl.h>
#include "..\libFluid2D\Fluid_kernel.cuh"

// Texture reference for reading velocity field
texture<float2, 2> g_texVeloc;//�����ڴ�������ֵ����ʱֻ֧��float��

__global__ void addForces_k(HBXDef::HbxCuDef::UserCalPrec2 * v, int dx, int dy, int spx, int spy, float fx, float fy, int r, size_t pitch)
{
	int idx = threadIdx.x;
	int idy = threadIdx.y;

	//��ά������ǰ�̶߳�Ӧ���ٶ���
	HBXDef::HbxCuDef::UserCalPrec2* fj = (HBXDef::HbxCuDef::UserCalPrec2*)((char*)v + (idy + spy)*pitch) + idx + spx;

	HBXDef::HbxCuDef::UserCalPrec2 tmpV = *fj;
	idx -= r;
	idy -= r;

	HBXDef::UserCalPrec s = 1.f / (1.f + idx * idx*idx*idx + idy * idy*idy*idy);
	tmpV.x += s * fx;
	tmpV.y += s * fy;
	*fj = tmpV;

}

__global__ void advectVelocity_k(	HBXDef::HbxCuDef::UserCalPrec2 * v, HBXDef::UserCalPrec * vx, HBXDef::UserCalPrec * vy, 
									int dx, int pdx, int dy, float dt, int lb)
{
	int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
	int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
	int p;

	float2 tmpV;	//��ʱ�ٶ������
	float2 pLoc;	//��ַ����������
	HBXDef::UserCalPrec vxterm, vyterm;

	//gtidx�Ǹ��߳�xλ�����ڵ��ٶ���
	if (gtidx < dx)
	{
		for (p = 0; p < lb; p++)
		{
			// fi�Ǹ��߳�yλ�����ڵ��ٶ���
			int fi = gtidy + p;
			if (fi < dy)
			{
				int fj = fi * pdx + gtidx;//���Ի��������
				tmpV = tex2D(g_texVeloc, (float)gtidx, (float)fi);
				pLoc.x = (gtidx + 0.5f) - (dt * tmpV.x * dx);
				pLoc.y = (fi + 0.5f) - (dt * tmpV.y * dy);
				tmpV = tex2D(g_texVeloc, pLoc.x, pLoc.y);
				vxterm = (HBXDef::UserCalPrec)tmpV.x;
				vyterm = (HBXDef::UserCalPrec)tmpV.y;
				vx[fj] = vxterm;
				vy[fj] = vyterm;
			}
		}
	}

}
