#include "cuFluid.h"
#include "..\libFluid2D\Fluid_kernel.cuh"
#include <hip/hip_runtime_api.h>

// �������ߴ�
size_t g_TexturePitch = 0;
hipArray* g_cuArray = nullptr;

void updateTexture(HBXDef::HbxCuDef::UserCalPrec2 *data, size_t wib, size_t h, size_t pitch)
{
	hipMemcpy2DToArray(g_cuArray, 0, 0, data, pitch, wib, h, hipMemcpyDeviceToDevice);
	getLastCudaError("hipMemcpy failed");
}


void addForces(	HBXDef::HbxCuDef::UserCalPrec2 * v, 
				int dx, int dy, 
				int spx, int spy,
				float fx, float fy, int r)
{
	dim3 tids(2*r+1, 2*r+1);
	addForces_k <<<1, tids >>> (v, dx, dy, spx, spy, fx, fy, r, g_TexturePitch);
	getLastCudaError("Func addForces_k failed");
}

void advectVelocity(HBXDef::HbxCuDef::UserCalPrec2 * v, 
					HBXDef::UserCalPrec * vx, HBXDef::UserCalPrec * vy,
					int dx, int pdx, int dy, HBXDef::UserCalPrec dt,
					int xTile, int yTile, 
					int xTidsPerTile, int yTidsPerTile,
					int _xDim, int _yDim)
{

	dim3 grids( (dx/xTile) + ((dx%xTile)?1:0) , (dy/yTile) + ((dy%yTile)?1:0) );
	dim3 blocks(xTidsPerTile, yTidsPerTile);

	updateTexture(v, _xDim * sizeof(HBXDef::HbxCuDef::UserCalPrec2), _yDim, g_TexturePitch);
	advectVelocity_k( v, vx, vy, dx, pdx, dy, dt, yTile/ yTidsPerTile);

	getLastCudaError("Func advectVelocity failed");
}

void diffuseProject(HBXDef::HbxCuDef::UserCalPrec2 * vx, HBXDef::HbxCuDef::UserCalPrec2 * vy, int dx, int dy, float dt, float visc)
{




	getLastCudaError("Func diffuseProject failed");
}
